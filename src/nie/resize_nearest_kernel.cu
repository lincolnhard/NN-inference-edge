#include "hip/hip_runtime.h"
#include <cassert>
#include <iostream>
#include "resize_nearest_kernel.h"
#include "common.h"


using namespace nvinfer1;

/// For checking if types are same
template <typename T, typename U>
struct is_same : std::false_type {};

template <typename T>
__device__ struct is_same<T, T> : std::true_type {};

template <typename T, typename U>
constexpr __device__ bool are_types_same() {
    return is_same<T, U>::value;
}

/// \brief CUDA kernel for calculating nearest neighbor resizing of stacked
/// images
///
/// \see https://en.wikipedia.org/wiki/Nearest-neighbor_interpolation
/// \see tensorflow/core/kernels/resize_nearest_neighbor_op_gpu.cu.cc
///
/// In this implementation there is one thread per output pixel
///
/// \tparam[in] T type of input tensors
/// \tparam[in] align_corners If true, scaling is (in-1)/(out-i), otherwise
///     in/out. True is normal computer vision / image processing nearest
///     resize, false is default for TensorFlow
/// \param[in] width_in Width of the original image
/// \param[in] height_in Height of the original image
/// \param[in] in Stacked input images in NHW order
/// \param[in] width_out Width of the resulting image
/// \param[in] height_out Height of the resulting image
/// \param[in] output_volume Volume of the output data
///     (width_out*height_out*layers)
/// \param[out] out Output data
template <typename T, bool align_corners>
__global__ void KernelResizeNearest(int const width_in, int const height_in,
                                    T const *in, int const width_out,
                                    int const height_out,
                                    uint32_t const output_volume,
                                    float const x_scale, float const y_scale,
                                    T *out) {
    // Calculate output pixel location
    uint32_t const idx_out = threadIdx.x + blockIdx.x * blockDim.x;

    // Make sure we do not over index mem
    if (idx_out > output_volume) {
        // This thread does not contribute
        return;
    }

    // These are whole integers, but we need them as floats
    float const out_x = static_cast<float>(idx_out % width_out);
    // Note that out_y is actually index in the whole block of images (across
    // layers)
    float const out_y = static_cast<float>(idx_out / width_out);

    // Y-index in 2D image
    float const in_y_img = (static_cast<int>(out_y) % height_out) * y_scale;

    // Input y is calculate little bit too complicated because of block shape
    // and thread indexing
    uint32_t const layer = out_y / height_out;

    // Calculate input pixel location (in floats)
    // Align corners is used tensorflows image_resizer_state.h to calculate the
    // scaling
    float const in_x = out_x * x_scale;

    // Calculate pixel coordinate
    // For the last row and column we want to make sure that we do not overindex
    int const in_xd0 =
        min(align_corners ? static_cast<int>(roundf(in_x))
                          : static_cast<int>(floorf(in_x)),
            width_in - 1);
    int const in_yd0 =
        min(align_corners ? static_cast<int>(roundf(in_y_img))
                          : static_cast<int>(floorf(in_y_img)),
            height_in - 1) + layer * height_in;

    // Possibilty to use __ldg
    T const py0x0 = in[in_yd0 * width_in + in_xd0];

    out[idx_out] = py0x0;
}

bool LauncherResizeNearest(int const num_batches, Dims const in_dims,
                            void const *data_in, Dims const size,
                            DataType const type, bool const align_corners,
                            hipStream_t stream, void *data_out) {
    assert(size.nbDims == 2);
    assert(in_dims.nbDims >= 2);

    // Calculate output tensor size
    Dims out_dims = in_dims;
    out_dims.d[out_dims.nbDims - 2] = size.d[0];
    out_dims.d[out_dims.nbDims - 1] = size.d[1];

    int const width_in = in_dims.d[in_dims.nbDims - 1];
    int const height_in = in_dims.d[in_dims.nbDims - 2];
    int const width_out = size.d[1];
    int const height_out = size.d[0];
    int const num_outputs = Volume(out_dims) * num_batches;

    // Kernel configuration
    int const threads_per_block = 128;
    int const blocks =
        (num_outputs + threads_per_block - 1) / threads_per_block;
    int const shared_mem = 0;

    // Calculate input pixel location (in floats)
    // Align corners is used tensorflows image_resizer_state.h to calculate the
    // scaling
    float const x_scale =
        align_corners ? (width_in - 1) / static_cast<float>(width_out - 1)
                      : width_in / static_cast<float>(width_out);
    float const y_scale =
        align_corners ? (height_in - 1) / static_cast<float>(height_out - 1)
                      : height_in / static_cast<float>(height_out);

    if (DataType::kFLOAT == type && !align_corners) {
        KernelResizeNearest<float,false>
            <<<blocks, threads_per_block, shared_mem, stream>>>(
                width_in, height_in, static_cast<float const *>(data_in),
                width_out, height_out, num_outputs, x_scale, y_scale,
                static_cast<float *>(data_out));
    } else if (DataType::kINT8 == type && !align_corners) {
        KernelResizeNearest<int8_t,false>
            <<<blocks, threads_per_block, shared_mem, stream>>>(
                width_in, height_in, static_cast<int8_t const *>(data_in),
                width_out, height_out, num_outputs, x_scale, y_scale,
                static_cast<int8_t *>(data_out));
    } else if (DataType::kFLOAT == type && align_corners) {
        KernelResizeNearest<float,true>
            <<<blocks, threads_per_block, shared_mem, stream>>>(
                width_in, height_in, static_cast<float const *>(data_in),
                width_out, height_out, num_outputs, x_scale, y_scale,
                static_cast<float *>(data_out));
    } else if (DataType::kINT8 == type && align_corners) {
        KernelResizeNearest<int8_t,true>
            <<<blocks, threads_per_block, shared_mem, stream>>>(
                width_in, height_in, static_cast<int8_t const *>(data_in),
                width_out, height_out, num_outputs, x_scale, y_scale,
                static_cast<int8_t *>(data_out));
    } else {
        std::cerr << "LauncherResizeNearest:Unsupported data type" << std::endl;
        return false;
    }

    auto err = hipStreamSynchronize(stream);
    if (hipSuccess != err) {
        std::cerr << "LauncherResizeNearest:Kernel launch failed: "
                  << hipGetErrorName(err) << std::endl;
        return false;
    }

    return true;
}
